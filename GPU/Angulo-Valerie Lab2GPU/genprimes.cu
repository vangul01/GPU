/*
 *  Last name: Angulo
 *  First name: Valerie
 *  Net ID: N14591814 
 *  if you use cuda2: nvcc -o heatdist -arch=sm_52 heatdist.cu
 *  if you use cuda5: nvcc -o heatdist -arch=sm_35 heatdist.cu
*/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

//define CUDA error checking
#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

//for cudamalloc
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if (hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString(err)); 
        exit(1);
    }
#endif
    return;
}

//for kernel
inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) { 
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
        exit(1);
    }
    // More careful checking. However, this will affect performance.
    //err = cudaDeviceSynchronize();
    //if(cudaSuccess != err) {
    //    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file, line, cudaGetErrorString(err));
    //    exit(1);
    //}
#endif
    return;
}

__global__ void FindPrimes (int* d_numbers, int N) {
	int tx = blockIdx.x * blockDim.x + threadIdx.x;	

	tx = min(tx, ((N+3)/2)); //decreases branch divergence: instead of if (tx <= ((N+1)/2)), exclusive N+3/2
	for (int i = tx+1; i < N-1; i++) { //check threads in positions tx+1->N-2
		if (d_numbers[tx] != 1) {
			if (d_numbers[i] % d_numbers[tx] == 0) {
				d_numbers[i] = 1;
			}
		}
	}
}

int main(int argc, char * argv[]) {

	if (argc != 2) {fprintf(stderr, "Missing file argument. Exiting\n"); exit(1);}
	int N = atoi(argv[argc-1]);
	int upper_bound = N-1;
	
	if (N <= 2 || N > 10000000) {fprintf(stderr, "N must be > 2 and <= 10,000,000\n"); exit(1);}
	
	int* number_arr = (int*)calloc(N-1, sizeof(int));
	if(!number_arr) {fprintf(stderr, " Cannot allocate the array\n"); exit(1);}

	//populate array with ints 2->N in array indices 0->N-2
	int index_int = 2;
	for (int i=0; i<upper_bound; i++) {
		number_arr[i] = index_int;
		index_int++; 
	}

/////////////////////////////////DEVICE CODE////////////////////////////
	//declare and allocate mem for device vars
	int* d_numbers;
	int size = (N-1) * sizeof(int);
	int blocks_per_grid = ((N-2)/1024) + 1;
	CudaSafeCall(hipMalloc((void**)&d_numbers, size));	

	//transfer data to device
	hipMemcpy(d_numbers, number_arr, size, hipMemcpyHostToDevice); 

	//setup kernel config
	dim3 dimGrid(blocks_per_grid, 1, 1); 
	dim3 dimBlock(1024, 1, 1);

	//call cuda kernel
	FindPrimes<<<dimGrid, dimBlock>>>(d_numbers, N);  
	CudaCheckError();

	//transfer results from d_numbers
	hipMemcpy(number_arr, d_numbers, size, hipMemcpyDeviceToHost);
///////////////////////////////////////////////////////////////////////// 

	FILE* fp;
	char filename[15]; //create string for file name
	sprintf(filename, "%d.txt", N);
	fp = freopen(filename, "w", stdout);

	//print array of primes to file
	for (int i = 0; i < upper_bound; i++) {
		if (number_arr[i] != 1) {
			printf("%d ", number_arr[i]);
		}
	}

	fclose(fp);
	hipFree(d_numbers);
	free(number_arr);
}

