/*
 *  Please write your name and net ID below
 *  
 *  Last name: Angulo
 *  First name: Valerie
 *  Net ID: N14591814
 * 
 */


/* 
 * This file contains the code for doing the heat distribution problem. 
 * You do not need to modify anything except starting  gpu_heat_dist() at the bottom
 * of this file.
 * In gpu_heat_dist() you can organize your data structure and the call to your
 * kernel(s) that you need to write too. 
 * 
 * You compile with:
 * 		nvcc -o heatdist -arch=sm_60 heatdist.cu   
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 

/* To index element (i,j) of a 2D array stored as 1D */
#define index(i, j, N)  ((i)*(N)) + (j)

//define CUDA error checking
#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

/*****************************************************************/

// Function declarations: Feel free to add any functions you want.
void  seq_heat_dist(float *, unsigned int, unsigned int);
void  gpu_heat_dist(float *, unsigned int, unsigned int);

//for cudamalloc
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
    return;
}

//for cuda kernel
inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
    return;
}

__global__ void CalculateHeat (float* d_temp, float* d_result, unsigned int N)
{
  int tx = blockIdx.x * blockDim.x + threadIdx.x;

  //compute only non-edge cases and put into d_result 
  //thread is within bounds of playground dimensions 
  if (tx < N*N) 
  { 
    if (tx > 0 && tx < N-1) //threads that are not on the edges
    { 
        for (int i = 1; i < N-1; i++) //i's that are not on the edges
        { 
          d_result[index(tx, i, N)] = (d_temp[index(tx-1, i, N)] + d_temp[index(tx+1, i, N)] 
            + d_temp[index(tx, i-1, N)] + d_temp[index(tx, i+1, N)])/4.0;
        }
    }
  }
}

/*****************************************************************/
/**** Do NOT CHANGE ANYTHING in main() function ******/

int main(int argc, char * argv[])
{
  unsigned int N; /* Dimention of NxN matrix */
  int type_of_device = 0; // CPU or GPU
  int iterations = 0;
  int i;
  
  /* The 2D array of points will be treated as 1D array of NxN elements */
  float * playground; 
  
  // to measure time taken by a specific part of the code 
  double time_taken;
  clock_t start, end;
  
  if(argc != 4)
  {
    fprintf(stderr, "usage: heatdist num  iterations  who\n");
    fprintf(stderr, "num = dimension of the square matrix (50 and up)\n");
    fprintf(stderr, "iterations = number of iterations till stopping (1 and up)\n");
    fprintf(stderr, "who = 0: sequential code on CPU, 1: GPU execution\n");
    exit(1);
  }
  
  type_of_device = atoi(argv[3]);
  N = (unsigned int) atoi(argv[1]);
  iterations = (unsigned int) atoi(argv[2]);
 
  
  /* Dynamically allocate NxN array of floats */
  playground = (float *)calloc(N*N, sizeof(float));
  if( !playground )
  {
   fprintf(stderr, " Cannot allocate the %u x %u array\n", N, N);
   exit(1);
  }
  
  /* Initialize it: calloc already initalized everything to 0 */
  // Edge elements to 70F
  for(i = 0; i < N; i++)
    playground[index(0,i,N)] = 70;
    
  for(i = 0; i < N; i++)
    playground[index(i,0,N)] = 70;
  
  for(i = 0; i < N; i++)
    playground[index(i,N-1, N)] = 70;
  
  for(i = 0; i < N; i++)
    playground[index(N-1,i,N)] = 70;
  
  // from (0,10) to (0,30) inclusive are 100F
  for(i = 10; i <= 30; i++)
    playground[index(0,i,N)] = 100;
  
   // from (n-1,10) to (n-1,30) inclusive are 150F
  for(i = 10; i <= 30; i++)
    playground[index(N-1,i,N)] = 150;
  
  if( !type_of_device ) // The CPU sequential version
  {  
    start = clock();
    seq_heat_dist(playground, N, iterations);
    end = clock();
  }
  else  // The GPU version
  {
     start = clock();
     gpu_heat_dist(playground, N, iterations); 
     end = clock();    
  }
  
  
  time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
  
  printf("Time taken for %s is %lf\n", type_of_device == 0? "CPU" : "GPU", time_taken);
  
  free(playground);
  
  return 0;

}


/*****************  The CPU sequential version (DO NOT CHANGE THAT) **************/
void  seq_heat_dist(float * playground, unsigned int N, unsigned int iterations)
{
  // Loop indices
  int i, j, k;
  int upper = N-1;
  
  // number of bytes to be copied between array temp and array playground
  unsigned int num_bytes = 0;
  
  float * temp; 
  /* Dynamically allocate another array for temp values */
  /* Dynamically allocate NxN array of floats */
  temp = (float *)calloc(N*N, sizeof(float));
  if( !temp )
  {
   fprintf(stderr, " Cannot allocate temp %u x %u array\n", N, N);
   exit(1);
  }
  
  num_bytes = N*N*sizeof(float);
  
  /* Copy initial array in temp */
  memcpy((void *)temp, (void *) playground, num_bytes);
  
  for( k = 0; k < iterations; k++)
  {
    /* Calculate new values and store them in temp */
    for(i = 1; i < upper; i++)
      for(j = 1; j < upper; j++)
	temp[index(i,j,N)] = (playground[index(i-1,j,N)] + 
	                      playground[index(i+1,j,N)] + 
			      playground[index(i,j-1,N)] + 
			      playground[index(i,j+1,N)])/4.0;
  
			      
   			      
    /* Move new values into old values */ 
    memcpy((void *)playground, (void *) temp, num_bytes);
  }
  
}

/***************** The GPU version: Write your code here *********************/
/* This function can call one or more kernels if you want ********************/
void  gpu_heat_dist(float * playground, unsigned int N, unsigned int iterations)
{
  //declare and allocate mem for device vars
  float* d_temp;
  float* d_result;
  float* pointer;
  int size = N * N * sizeof(float);
  int blocks_per_grid = ((N*N)/512) + 1; //enough threads for each point in playground
  CudaSafeCall(hipMalloc((void**)&d_temp, size));
  CudaSafeCall(hipMalloc((void**)&d_result, size));

  //transfer data to device, both have playground now
  hipMemcpy(d_temp, playground, size, hipMemcpyHostToDevice);
  hipMemcpy(d_result, playground, size, hipMemcpyHostToDevice);

  //setup kernel config
  dim3 dimGrid(blocks_per_grid, 1, 1); //based on dimensions of playground
  dim3 dimBlock(512, 1, 1); //512 threads per block 

  //start of iteration loop and kernel execution
  for(int it = 0; it < iterations; ++it) {
    //call cuda kernel to update solution using old solution, put updated solution in d_result
    CalculateHeat<<<dimGrid, dimBlock>>>(d_temp, d_result, N);  

    //swap pointer to make temp get result, result becomes old 
    pointer = d_temp;
    d_temp = d_result;
    d_result = pointer;
  }

  //transfer results from d_temp that holds results to playground
  hipMemcpy(playground, d_temp, size, hipMemcpyDeviceToHost);

  //free mem
  hipFree(d_temp);
  hipFree(d_result);
}
